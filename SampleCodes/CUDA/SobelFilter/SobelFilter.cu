#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <vector>
#include <numeric>
#include <algorithm>
#include <cmath>

#include "hip/hip_vector_types.h"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void sobel(hipSurfaceObject_t output, hipTextureObject_t input)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    float4 p[3*3];
    for(int dy = -1; dy <= 1; dy += 1)
    {
        for(int dx = -1; dx <= 1; dx += 1)
        {
            p[(dy+1) * 3 + (dx+1)] = tex2D<float4>(input, x+dx, y+dy);
        }
    }

    const float4 gradient_x = p[0*3+0] - p[0*3+2] + 2.0f * (p[1*3+0] - p[1*3+2]) + p[2*3+0] - p[2*3+2];
    const float4 gradient_y = p[0*3+0] - p[2*3+0] + 2.0f * (p[0*3+1] - p[2*3+1]) + p[0*3+2] - p[2*3+2];

    const float gradient = max(0.0f, min(1.0f, 0.25f * sqrt( dot(gradient_x, gradient_x) + dot(gradient_y, gradient_y) ) ) );
    surf2Dwrite(float4{gradient, gradient, gradient, 1.0f}, output, x * sizeof(float4), y, hipSurfaceBoundaryMode::hipBoundaryModeZero);
}

struct rawcolor{ unsigned char r, g, b; };

int main()
{
    hipError_t err = hipSuccess;
    
    // Using the implicitely selected first cuda device:
    hipDeviceProp_t prop;
    err = hipGetDeviceProperties(&prop, 0);
    if(err != hipSuccess){ std::cout << "Error getting device properties for device 0: " << hipGetErrorString(err) << "\n"; return -1; }
    std::cout << "Selected device name: " << prop.name << "\n";
    
    // Actual program logic:
    // Read input image:
    int w = 0; // width
    int h = 0; // height
    int ch = 0; // number of components

    std::string input_filename = "Valve_original.png";
    rawcolor* data0 = reinterpret_cast<rawcolor*>(stbi_load(input_filename.c_str(), &w, &h, &ch, 3 /* we expect 3 components */));
    if(!data0)
    {
        std::cout << "Error: could not open input file: " << input_filename << "\n";
        return -1;
    }
    else
    {
        std::cout << "Image (" << input_filename << ") opened successfully. Width x Height x Components = " << w << " x " << h << " x " << ch << "\n";
    }

    std::vector<float4> input (w*h);
    std::vector<float4> output(w*h);

    std::transform(data0, data0+w*h, input.begin(), [](rawcolor c){ return float4{c.r/255.0f, c.g/255.0f, c.b/255.0f, 1.0f}; } );
    
    // Create cuda texture and surface objects:
    // Channel layout of data:
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
       
    // Allocate data:
    hipArray* arr_input = nullptr;
    hipArray* arr_output = nullptr;

    err = hipMallocArray(&arr_input, &channelDesc, w, h);
    if( err != hipSuccess){ std::cout << "Error allocating CUDA memory arr_input: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipMallocArray(&arr_output, &channelDesc, w, h);
    if( err != hipSuccess){ std::cout << "Error allocating CUDA memory arr_output: " << hipGetErrorString(err) << "\n"; return -1; }

    // Upload data to device:
    err = hipMemcpyToArray(arr_input,  0, 0, input.data(), w*h*sizeof(float4), hipMemcpyHostToDevice);
    if( err != hipSuccess){ std::cout << "Error copying memory to device: " << hipGetErrorString(err) << "\n"; return -1; }
        
    // Specify texture resource description:
    hipResourceDesc resdescInput{};
    resdescInput.resType = hipResourceTypeArray;
    resdescInput.res.array.array = arr_input;

    // Specify texture description:
    hipTextureDesc texDesc{};
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.addressMode[1]   = hipAddressModeBorder;
    texDesc.filterMode       = hipFilterModePoint;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // Create texture object
    hipTextureObject_t texture = 0;
    err = hipCreateTextureObject(&texture,  &resdescInput,  &texDesc, nullptr);
    if( err != hipSuccess){ std::cout << "Error creating texture object: " << hipGetErrorString(err) << "\n"; return -1; }
        
    // Create the surface object that will hold the output:
    hipSurfaceObject_t surface;
    // Specify surface resource description:
    hipResourceDesc resdescOutput{};
    resdescOutput.resType = hipResourceTypeArray;
    resdescOutput.res.array.array = arr_output;

    err = hipCreateSurfaceObject(&surface, &resdescOutput);
    if( err != hipSuccess){ std::cout << "Error creating surface object: " << hipGetErrorString(err) << "\n"; return -1; }
    
    // Launch kernel:
    {
        const int block = 16;
        dim3 grid_size( static_cast<int>( std::ceil(w*1.0f/block) ), static_cast<int>( std::ceil(w*1.0f/block) ) );
	    dim3 block_size( block, block );
	    sobel<<<grid_size, block_size>>>(surface, texture);

	    err = hipGetLastError();
	    if(err != hipSuccess){ std::cout << "CUDA error in kernel call: " << hipGetErrorString(err) << "\n"; return -1; }
    }

    // Copy back results (implicitely synchronizes on the default stream that we are using):
    err = hipMemcpyFromArray(output.data(), arr_output, 0, 0, w*h*sizeof(float4), hipMemcpyDeviceToHost);
    if(err != hipSuccess){ std::cout << "Error copying memory to host: " << hipGetErrorString(err) << "\n"; return -1; }

    // Write out image:
    {
        std::vector<rawcolor> tmp(w*h);
        std::transform(output.cbegin(), output.cend(), tmp.begin(),
            [](float4 c){ return rawcolor{  (unsigned char)(c.x*255.0f),
                                            (unsigned char)(c.y*255.0f),
                                            (unsigned char)(c.z*255.0f) }; } );

        const std::string output_filename = "result.jpg";
        int res = stbi_write_jpg(output_filename.c_str(), w, h, ch, tmp.data(), 100);
        if(res == 0)
        {
            std::cout << "Error writing output to file " << output_filename << "\n";
        }else{ std::cout << "Output written to file " << output_filename << "\n"; }
    }

    // Clean-up:
	err = hipDestroySurfaceObject(surface);
    if(err != hipSuccess){ std::cout << "Error destroying surface object: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipDestroyTextureObject(texture);
    if(err != hipSuccess){ std::cout << "Error destroying texture object: " << hipGetErrorString(err) << "\n"; return -1; }

	err = hipFreeArray( arr_output );
	if(err != hipSuccess){ std::cout << "Error freeing array arr_output: " << hipGetErrorString(err) << "\n"; return -1; }

    err = hipFreeArray( arr_input );
	if(err != hipSuccess){ std::cout << "Error freeing array arr_input: " << hipGetErrorString(err) << "\n"; return -1; }

    stbi_image_free(data0);

	return 0;
}