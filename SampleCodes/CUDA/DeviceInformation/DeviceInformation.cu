
#include <hip/hip_runtime.h>
#include <iostream>

int main()
{
    hipError_t err = hipSuccess;

	int device_count = 0;

    err = hipGetDeviceCount(&device_count);
    if( err != hipSuccess){ std::cout << "Error getting device count: " << hipGetErrorString(err) << "\n"; return -1; }
    
    std::cout << "There are " << device_count << " device(s)\n";
    
    for(int d = 0; d < device_count; ++d)
    {
        hipDeviceProp_t prop;
        err = hipGetDeviceProperties(&prop, d);
        if( err != hipSuccess){ std::cout << "Error getting device properties for device " << d << ": " << hipGetErrorString(err) << "\n"; continue; }
        std::cout << "Device " << d << " name:               " << prop.name << "\n";
        std::cout << "Device " << d << " global memory size: " << prop.totalGlobalMem/1024/1024 << " MiB\n";
    }

	return 0;
}